#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

__global__ void calculate(int height, int width, int max_iterations, double *red_pixels, double *green_pixels, double *blue_pixels);

int main(int argc, char** argv) {
	// Rozmery obrazka
	const int HEIGHT = 800;
	const int WIDTH = 800;
	// Maximalny pocet iteracii v ktorych bude prebiehat vypocet
	const int MAX_ITERATIONS = 100000;

	int total_pixels = HEIGHT*WIDTH;
	hipEvent_t start_time, end_time;
	float result_time = 0;

	// CPU alokacia farebnych poli
	double * cpu_red_array = (double *) malloc(total_pixels * sizeof(double));
	double * cpu_green_array = (double *) malloc(total_pixels * sizeof(double));
	double * cpu_blue_array = (double *) malloc(total_pixels * sizeof(double));

	// GPU alokacia farebnych poli
	double *device_red_array, *device_green_array, *device_blue_array;

	hipMalloc((void**)&device_red_array, total_pixels * sizeof(double));
	hipMalloc((void**)&device_green_array, total_pixels * sizeof(double));
	hipMalloc((void**)&device_blue_array, total_pixels * sizeof(double));

	// Zaciatok trvania vypoctu
	srand(time(NULL));
	hipEventCreate(&start_time);
	hipEventCreate(&end_time);
	hipEventRecord(start_time, 0);

	// Vypocet na CUDE, grid ma 256 thread blokov, kde kazdy z nich ma 256 threadov
	calculate<<<256, 256>>>(HEIGHT, WIDTH, MAX_ITERATIONS, device_red_array, device_green_array, device_blue_array);

	// Koniec trvania vypoctu
	hipEventRecord(end_time, 0);
	hipEventSynchronize(end_time);
	hipEventElapsedTime(&result_time, start_time, end_time);

	printf("Vypocer trval: %3.3f sekund.\n", result_time/1000);

	//Presun dat z device do CPU
	hipMemcpy(cpu_red_array, device_red_array, total_pixels * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(cpu_green_array, device_green_array, total_pixels * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(cpu_blue_array, device_blue_array, total_pixels * sizeof(double), hipMemcpyDeviceToHost);

	// Zapis vysledkov do suboru
	FILE *file = fopen("output.txt", "w");

	for (int i = 0; i <total_pixels; i++) {
		fprintf(file, "%lf,%lf,%lf\n", cpu_red_array[i], cpu_green_array[i], cpu_blue_array[i]);
	}

	fclose(file);

	// Uvolnenie zdrojov
	hipFree(device_red_array);
	hipFree(device_green_array);
	hipFree(device_blue_array);
	free(cpu_red_array);
	free(cpu_green_array);
	free(cpu_blue_array);

	hipEventDestroy(start_time);
	hipEventDestroy(end_time);
	hipDeviceReset();

	return 0;
}

__global__ void calculate(int height, int width, int max_iterations, double *red_pixels, double *green_pixels, double *blue_pixels) {
	int id = blockIdx.x * blockDim.x + threadIdx.x; // generating unique thread index
	int total_pixels = height * width;

	while (id < total_pixels) {
		int c = id / width;
		int r = id % height;

		int currentIndex = c + (r * width);

		double x_axis_offset = -(width)/1.4;
		double y_axis_offset = (height)/2.0;

		double c_real = (x_axis_offset + c)/300;
		double c_img = (y_axis_offset - r)/300;
		double z_real = 0.0, z_imag = 0.0, z_real_tmp = 0.0, z_img_tmp = 0.0;
		double absolut = 0.0;

		int iter = 0;

		while (iter < max_iterations && absolut <= 4.0) {
			z_real = z_real_tmp*z_real_tmp - z_img_tmp*z_img_tmp + c_real;
			z_imag = 2.0*z_real_tmp*z_img_tmp + c_img;
			absolut = z_real*z_real + z_imag*z_imag;
			z_real_tmp = z_real;
			z_img_tmp = z_imag;

			iter++;
		}

		if (iter == max_iterations) {
			// pixel bude zafarbeny na cierno
			red_pixels[currentIndex] = 0.0;
			green_pixels[currentIndex] = 0.0;
			blue_pixels[currentIndex] = 0.0;
		}
		else {
			// pixel bude zvyrazneny
			red_pixels[currentIndex] = pow(((double) iter)/((double)max_iterations), 0.25);
			green_pixels[currentIndex] = 1.0;

			if (iter < max_iterations) {
				blue_pixels[currentIndex] = 1.0;
			}else {
				blue_pixels[currentIndex] = 0.0;
			}
		}
		id += blockDim.x * gridDim.x; //grid-type loop, specificke pre CUDU
	}
}
